#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;
#include "timer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

int n,m,*edg,*degree,*startNode;
thrust::host_vector<thrust::pair<int,int> > stEdges;
int md,*dedg,*dstartNode,*dresult;
int threads_per_block = 512,blocks_per_grid = 32;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void numTri(int m,const int * __restrict__ edg,const int * __restrict__ startNode,int * result) {
    int t = blockDim.x * blockIdx.x + threadIdx.x,ret = 0;
    int numThreads = gridDim.x * blockDim.x; 
    if(t < m) {
    	int rem = m % numThreads,count = m/numThreads,st,en;
	    if(t < rem) {
	        st = t * (count + 1);
	        en = st + count;
	    }
	    else {
	        st = t*count + rem;
	        en = st + count - 1 ;
	    }
    	for(int i=st;i<=en;i++) {
    		int u = edg[i],v = edg[m+i];
			int su = startNode[u],eu = startNode[u+1]-1; int sv = startNode[v],ev = startNode[v+1]-1;
			while(su <= eu and sv <= ev) {
				int diff = edg[su+m]-edg[sv+m];
				if(diff == 0) {
					su++; sv++; ret++;
				}
				else if(diff > 0) sv++;
				else su++;
			}
	    }
    }
    result[t] = ret;
}

void setupDeviceMemory() {
	int sizeEdg = 2*m * sizeof(int);
	int tem = threads_per_block * blocks_per_grid * sizeof(int);
	gpuErrchk(hipMalloc(&dedg,sizeEdg));
	gpuErrchk(hipMalloc(&dstartNode,(n+1) * sizeof(int)));
	gpuErrchk(hipMalloc(&dresult,tem));  
   	gpuErrchk(hipMemcpy(dedg,edg,sizeEdg,hipMemcpyHostToDevice));
   	gpuErrchk(hipMemcpy(dstartNode,startNode,(n+1) * sizeof(int),hipMemcpyHostToDevice));
}

void freeDeviceMemory() {
	free(edg); free(degree); free(startNode);
	gpuErrchk(hipFree(dedg)); gpuErrchk(hipFree(dstartNode)); gpuErrchk(hipFree(dresult));
}

int main() {
	scanf("%d %d",&n,&m);
	int sizeVer = n * sizeof(int),sizeEdg = 2*m * sizeof(int);
	edg = (int *) malloc(sizeEdg);
	degree = (int *) malloc(sizeVer);
	startNode = (int *) malloc((n+1) * sizeof(int));
	for(int i=0;i<n;i++) {
		degree[i] = 0;
	}
	for(int i = 0 ; i < m ; i++) {
		int node1,node2;
		scanf("%d %d",&node1,&node2);
		stEdges.push_back(make_pair(node1,node2));
		degree[node1]++; degree[node2]++;
	}
	for(int i = 0 ;i < stEdges.size(); i++) {
		if(degree[stEdges[i].first] > degree[stEdges[i].second]) {
			swap(stEdges[i].first,stEdges[i].second);
		}
	}
	thrust::device_vector<thrust::pair<int,int> > dEdg = stEdges;
	thrust::sort(dEdg.begin(),dEdg.end());
	stEdges = dEdg;

	int pres = 0;
	for(int i = 0 ;i < stEdges.size(); i++) {
		edg[i] = stEdges[i].first; edg[i+m] = stEdges[i].second;
		while(pres <= stEdges[i].first) {
			startNode[pres] = i;
			pres++;
		}
	}
	while(pres <= n) {
		startNode[pres] = m;
		pres++;
	}

	double start,finish;
	GET_TIME(start);
	setupDeviceMemory();
	numTri<<<blocks_per_grid,threads_per_block>>>(m,dedg,dstartNode,dresult);
	hipDeviceSynchronize();
	thrust::device_ptr<int> dptr(dresult);
	int  result = thrust::reduce(dptr,dptr+(threads_per_block*blocks_per_grid));
	cout << result << "\n";
	GET_TIME(finish);
	hipDeviceSynchronize();
	freeDeviceMemory();
	printf("Elapsed time = %e seconds\n",finish - start);
}